#include "hip/hip_runtime.h"
#include "ObjectiveFunction.cuh"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <chrono>
#include <iostream>
#include <iomanip>
#include <vector>
#include <algorithm>

struct Particle {
    int position[NUM_CITIES];
    double velocity[NUM_CITIES];
    int bestPosition[NUM_CITIES];
    double bestFitness;
};

__device__ double calculateDistance(int* position) {
    double distance = 0.0;
    for (int i = 0; i < NUM_CITIES - 1; i++) {
        int city1 = position[i];
        int city2 = position[i + 1];
        distance += distances[city1][city2];
    }
    int lastCity = position[NUM_CITIES - 1];
    int firstCity = position[0];
    distance += distances[lastCity][firstCity];
    return distance;
}

__device__ void updateBestFitness(Particle* p, int* globalBestPosition, double* globalBestFitness) {
    double fitness = calculateDistance(p->position);
    if (fitness < p->bestFitness) {
        p->bestFitness = fitness;
        for (int i = 0; i < NUM_CITIES; i++) {
            p->bestPosition[i] = p->position[i];
        }
    }
    if (fitness < *globalBestFitness) {
        *globalBestFitness = fitness;
        for (int i = 0; i < NUM_CITIES; i++) {
            globalBestPosition[i] = p->position[i];
        }
    }
}

__global__ void initializeParticles(Particle* particles, int* globalBestPosition, double* globalBestFitness, hiprandState* state) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < NUM_PARTICLES) {
        Particle* p = &particles[tid];
        hiprandState* s = &state[tid];
        hiprand_init(clock64(), tid, 0, s);
        for (int i = 0; i < NUM_CITIES; i++) {
            p->position[i] = i;
        }
        for (int i = NUM_CITIES - 1; i > 0; i--) {
            int j = hiprand(s) % (i + 1);
            int temp = p->position[i];
            p->position[i] = p->position[j];
            p->position[j] = temp;
        }
        for (int i = 0; i < NUM_CITIES; i++) {
            p->velocity[i] = hiprand_uniform_double(s) * 2.0 - 1.0;
            p->bestPosition[i] = p->position[i];
        }
        p->bestFitness = calculateDistance(p->position);
        updateBestFitness(p, globalBestPosition, globalBestFitness);
    }
}

__global__ void updateParticles(Particle* particles, int* globalBestPosition, double* globalBestFitness, hiprandState* state) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < NUM_PARTICLES) {
        Particle* p = &particles[tid];
        hiprandState* s = &state[tid];
        for (int i = 0; i < NUM_CITIES; i++) {
            double r1 = hiprand_uniform_double(s);
            double r2 = hiprand_uniform_double(s);
            p->velocity[i] = INERTIA_WEIGHT * p->velocity[i] +
                             COGNITIVE_WEIGHT * r1 * (p->bestPosition[i] - p->position[i]) +
                             SOCIAL_WEIGHT * r2 * (globalBestPosition[i] - p->position[i]);
        }
        int newPosition[NUM_CITIES];
        for (int i = 0; i < NUM_CITIES; i++) {
            newPosition[i] = p->position[i];
        }
        for (int i = 0; i < NUM_CITIES; i++) {
            int swapIndex = (static_cast<int>(p->velocity[i]) + NUM_CITIES) % NUM_CITIES;
            int temp = newPosition[i];
            newPosition[i] = newPosition[swapIndex];
            newPosition[swapIndex] = temp;
        }
        bool isValid = true;
        for (int i = 0; i < NUM_CITIES; i++) {
            bool found = false;
            for (int j = 0; j < NUM_CITIES; j++) {
                if (newPosition[j] == i) {
                    found = true;
                    break;
                }
            }
            if (!found) {
                isValid = false;
                break;
            }
        }
        if (isValid) {
            for (int i = 0; i < NUM_CITIES; i++) {
                p->position[i] = newPosition[i];
            }
            updateBestFitness(p, globalBestPosition, globalBestFitness);
        }
    }
}

void runPSO(Particle* particles, int* globalBestPosition, double* globalBestFitness, hiprandState* state) {
    dim3 block(BLOCK_SIZE);
    dim3 grid((NUM_PARTICLES + block.x - 1) / block.x);
    for (int iter = 0; iter < MAX_ITERATIONS; iter++) {
        updateParticles<<<grid, block>>>(particles, globalBestPosition, globalBestFitness, state);
        hipDeviceSynchronize();
    }
}

void printResults(int* globalBestPosition, double globalBestFitness, double executionTime) {
    std::cout << "Best Path: ";
    for (int i = 0; i < NUM_CITIES; i++) {
        std::cout << globalBestPosition[i] << " ";
    }
    std::cout << std::endl;
    std::cout << "Best Distance: " << globalBestFitness << std::endl;
    std::cout << std::fixed << std::setprecision(2);
    std::cout << "Execution Time: " << executionTime << " milliseconds" << std::endl;
}

int main() {
    Particle* particles;
    int* globalBestPosition;
    double* globalBestFitness;
    hiprandState* state;
    hipMalloc(&particles, NUM_PARTICLES * sizeof(Particle));
    hipMalloc(&globalBestPosition, NUM_CITIES * sizeof(int));
    hipMalloc(&globalBestFitness, sizeof(double));
    hipMalloc(&state, NUM_PARTICLES * sizeof(hiprandState));
    double initialFitness = INFINITY;
    hipMemcpy(globalBestFitness, &initialFitness, sizeof(double), hipMemcpyHostToDevice);
    auto start = std::chrono::high_resolution_clock::now();
    initializeParticles<<<(NUM_PARTICLES + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(particles, globalBestPosition, globalBestFitness, state);
    hipDeviceSynchronize();
    runPSO(particles, globalBestPosition, globalBestFitness, state);
    auto end = std::chrono::high_resolution_clock::now();
    double executionTime = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    int* hostGlobalBestPosition = new int[NUM_CITIES];
    double hostGlobalBestFitness;
    hipMemcpy(hostGlobalBestPosition, globalBestPosition, NUM_CITIES * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&hostGlobalBestFitness, globalBestFitness, sizeof(double), hipMemcpyDeviceToHost);
    printResults(hostGlobalBestPosition, hostGlobalBestFitness, executionTime);
    delete[] hostGlobalBestPosition;
    hipFree(particles);
    hipFree(globalBestPosition);
    hipFree(globalBestFitness);
    hipFree(state);
    hipDeviceReset();
    return 0;
}
