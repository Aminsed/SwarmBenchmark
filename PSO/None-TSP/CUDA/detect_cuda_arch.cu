#include <iostream>
#include <hip/hip_runtime.h>

int main() {
    int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);

    if (error_id != hipSuccess) {
        std::cerr << "Error: hipGetDeviceCount failed with error: " << hipGetErrorString(error_id) << std::endl;
        return 1;
    }

    if (deviceCount == 0) {
        std::cerr << "Error: No CUDA-capable devices found." << std::endl;
        return 1;
    }

    int device = 0;
    error_id = hipSetDevice(device);

    if (error_id != hipSuccess) {
        std::cerr << "Error: hipSetDevice failed with error: " << hipGetErrorString(error_id) << std::endl;
        return 1;
    }

    hipDeviceProp_t deviceProp;
    error_id = hipGetDeviceProperties(&deviceProp, device);

    if (error_id != hipSuccess) {
        std::cerr << "Error: hipGetDeviceProperties failed with error: " << hipGetErrorString(error_id) << std::endl;
        return 1;
    }

    std::cout << "sm_" << deviceProp.major << deviceProp.minor << std::endl;

    return 0;
}
