#include "hip/hip_runtime.h"
// ParticleSwarmOptimization.cu
#include "ObjectiveFunction.cuh"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <chrono>
#include <iostream>
#include <iomanip>


struct Particle {
    double position[DIMENSIONS];
    double velocity[DIMENSIONS];
    double bestPosition[DIMENSIONS];
    double bestFitness;
};

__device__ void updateBestFitness(Particle* p, double* globalBestPosition, double* globalBestFitness) {
    double fitness = objectiveFunction(p->position);

    if (fitness < p->bestFitness) {
        p->bestFitness = fitness;
        for (int i = 0; i < DIMENSIONS; i++) {
            p->bestPosition[i] = p->position[i];
        }
    }

    if (fitness < *globalBestFitness) {
        *globalBestFitness = fitness;
        for (int i = 0; i < DIMENSIONS; i++) {
            globalBestPosition[i] = p->position[i];
        }
    }
}

__global__ void initializeParticles(Particle* particles, double* globalBestPosition, double* globalBestFitness, hiprandState* state) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < NUM_PARTICLES) {
        Particle* p = &particles[tid];
        hiprandState* s = &state[tid];
        hiprand_init(clock64(), tid, 0, s);

        for (int i = 0; i < DIMENSIONS; i++) {
            p->position[i] = hiprand_uniform_double(s) * 10.0 - 5.0;
            p->velocity[i] = hiprand_uniform_double(s) * 2.0 - 1.0;
            p->bestPosition[i] = p->position[i];
        }
        p->bestFitness = INFINITY;

        updateBestFitness(p, globalBestPosition, globalBestFitness);
    }
}

__global__ void updateParticles(Particle* particles, double* globalBestPosition, double* globalBestFitness, hiprandState* state) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < NUM_PARTICLES) {
        Particle* p = &particles[tid];
        hiprandState* s = &state[tid];

        for (int i = 0; i < DIMENSIONS; i++) {
            double r1 = hiprand_uniform_double(s);
            double r2 = hiprand_uniform_double(s);

            p->velocity[i] = INERTIA_WEIGHT * p->velocity[i] +
                             COGNITIVE_WEIGHT * r1 * (p->bestPosition[i] - p->position[i]) +
                             SOCIAL_WEIGHT * r2 * (globalBestPosition[i] - p->position[i]);

            p->position[i] += p->velocity[i];
        }

        updateBestFitness(p, globalBestPosition, globalBestFitness);
    }
}

void runPSO(Particle* particles, double* globalBestPosition, double* globalBestFitness, hiprandState* state) {
    dim3 block(BLOCK_SIZE);
    dim3 grid((NUM_PARTICLES + block.x - 1) / block.x);

    for (int iter = 0; iter < MAX_ITERATIONS; iter++) {
        updateParticles<<<grid, block>>>(particles, globalBestPosition, globalBestFitness, state);
        hipDeviceSynchronize();
    }
}

void printResults(double* globalBestPosition, double globalBestFitness, double executionTime) {
    std::cout << std::fixed << std::setprecision(10);
    if (DIMENSIONS == 1) {
        std::cout << "Global Best Position: " << globalBestPosition[0] << std::endl;
    } else {
        std::cout << "Global Best Position: (";
        for (int i = 0; i < DIMENSIONS; i++) {
            std::cout << globalBestPosition[i];
            if (i < DIMENSIONS - 1) {
                std::cout << ", ";
            }
        }
        std::cout << ")" << std::endl;
    }
    std::cout << "Global Best Value: " << globalBestFitness << std::endl;
    std::cout << std::fixed << std::setprecision(2);
    std::cout << "Execution Time: " << executionTime << " milliseconds" << std::endl;
}

int main() {
    Particle* particles;
    double* globalBestPosition;
    double* globalBestFitness;
    hiprandState* state;

    hipMalloc(&particles, NUM_PARTICLES * sizeof(Particle));
    hipMalloc(&globalBestPosition, DIMENSIONS * sizeof(double));
    hipMalloc(&globalBestFitness, sizeof(double));
    hipMalloc(&state, NUM_PARTICLES * sizeof(hiprandState));

    double initialFitness = INFINITY;
    hipMemcpy(globalBestFitness, &initialFitness, sizeof(double), hipMemcpyHostToDevice);

    auto start = std::chrono::high_resolution_clock::now();

    initializeParticles<<<(NUM_PARTICLES + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(particles, globalBestPosition, globalBestFitness, state);
    hipDeviceSynchronize();

    runPSO(particles, globalBestPosition, globalBestFitness, state);

    auto end = std::chrono::high_resolution_clock::now();
    double executionTime = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();

    double hostGlobalBestPosition[DIMENSIONS];
    double hostGlobalBestFitness;
    hipMemcpy(hostGlobalBestPosition, globalBestPosition, DIMENSIONS * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(&hostGlobalBestFitness, globalBestFitness, sizeof(double), hipMemcpyDeviceToHost);

    printResults(hostGlobalBestPosition, hostGlobalBestFitness, executionTime);

    hipFree(particles);
    hipFree(globalBestPosition);
    hipFree(globalBestFitness);
    hipFree(state);

    hipDeviceReset();

    return 0;
}
