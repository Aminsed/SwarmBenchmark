#include "Swarm.h"
#include "ObjectiveFunction.h"
#include <iostream>
#include <hip/hip_runtime.h>

int main() {
    size_t swarmSize = 100;
    double searchSpaceMin = -5.12, searchSpaceMax = 5.12;
    int maxIterations = 1000;

    // Allocate device memory for global best position
    double* d_globalBestPosition;
    hipMalloc(&d_globalBestPosition, 2 * sizeof(double));

    // Create a swarm with the specified parameters
    Swarm swarm(swarmSize, searchSpaceMin, searchSpaceMax, ObjectiveFunction::rastrigin);

    // Initialize the swarm
    swarm.initialize();

    // Run the PSO algorithm
    swarm.optimize(maxIterations, d_globalBestPosition);

    // Copy global best position from device to host
    double globalBestPosition[2];
    hipMemcpy(globalBestPosition, d_globalBestPosition, 2 * sizeof(double), hipMemcpyDeviceToHost);

    // Print the global best position and value
    std::cout << "Global Best Position: (" << globalBestPosition[0] << ", " << globalBestPosition[1] << ")" << std::endl;
    std::cout << "Global Best Value: " << swarm.getGlobalBestValue() << std::endl;

    // Free device memory
    hipFree(d_globalBestPosition);

    return 0;
}
