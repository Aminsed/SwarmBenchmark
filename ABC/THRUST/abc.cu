#include "hip/hip_runtime.h"
#include "ObjectiveFunction.cuh"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/extrema.h>
#include <chrono>
#include <iostream>
#include <iomanip>

struct FoodSource {
    double position[DIMENSIONS];
    double fitness;
    int trialCount;
};

__device__ void updateFitness(FoodSource* fs) {
    fs->fitness = objectiveFunction(fs->position);
}

__global__ void initializeFoodSources(FoodSource* foodSources, hiprandState* state) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < NUM_FOOD_SOURCES) {
        FoodSource* fs = &foodSources[tid];
        hiprandState* s = &state[tid];
        hiprand_init(clock64(), tid, 0, s);
        for (int i = 0; i < DIMENSIONS; i++) {
            fs->position[i] = hiprand_uniform_double(s) * 10.0 - 5.0;
        }
        updateFitness(fs);
        fs->trialCount = 0;
    }
}

__global__ void sendEmployedBees(FoodSource* foodSources, hiprandState* state) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < NUM_FOOD_SOURCES) {
        FoodSource* fs = &foodSources[tid];
        hiprandState* s = &state[tid];
        int j = hiprand_uniform(s) * DIMENSIONS;
        double phi = hiprand_uniform_double(s) * 2.0 - 1.0;
        FoodSource newFs = *fs;
        newFs.position[j] += phi * (newFs.position[j] - foodSources[(int)(hiprand_uniform(s) * NUM_FOOD_SOURCES)].position[j]);
        updateFitness(&newFs);
        if (newFs.fitness < fs->fitness) {
            *fs = newFs;
            fs->trialCount = 0;
        } else {
            fs->trialCount++;
        }
    }
}

__global__ void sendOnlookerBees(FoodSource* foodSources, hiprandState* state) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < NUM_FOOD_SOURCES) {
        hiprandState* s = &state[tid];
        double probabilities[NUM_FOOD_SOURCES];
        double maxFitness = foodSources[0].fitness;
        for (int i = 1; i < NUM_FOOD_SOURCES; i++) {
            if (foodSources[i].fitness > maxFitness) {
                maxFitness = foodSources[i].fitness;
            }
        }
        double fitnessSum = 0.0;
        for (int i = 0; i < NUM_FOOD_SOURCES; i++) {
            probabilities[i] = (0.9 * (foodSources[i].fitness / maxFitness)) + 0.1;
            fitnessSum += probabilities[i];
        }
        double r = hiprand_uniform_double(s) * fitnessSum;
        double cumulativeProbability = 0.0;
        int selectedIndex = 0;
        for (int i = 0; i < NUM_FOOD_SOURCES; i++) {
            cumulativeProbability += probabilities[i];
            if (r <= cumulativeProbability) {
                selectedIndex = i;
                break;
            }
        }
        FoodSource* fs = &foodSources[selectedIndex];
        int j = hiprand_uniform(s) * DIMENSIONS;
        double phi = hiprand_uniform_double(s) * 2.0 - 1.0;
        FoodSource newFs = *fs;
        newFs.position[j] += phi * (newFs.position[j] - foodSources[(int)(hiprand_uniform(s) * NUM_FOOD_SOURCES)].position[j]);
        updateFitness(&newFs);
        if (newFs.fitness < fs->fitness) {
            *fs = newFs;
            fs->trialCount = 0;
        } else {
            fs->trialCount++;
        }
    }
}

__global__ void sendScoutBees(FoodSource* foodSources, hiprandState* state) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < NUM_FOOD_SOURCES) {
        FoodSource* fs = &foodSources[tid];
        if (fs->trialCount >= LIMIT) {
            hiprandState* s = &state[tid];
            for (int i = 0; i < DIMENSIONS; i++) {
                fs->position[i] = hiprand_uniform_double(s) * 10.0 - 5.0;
            }
            updateFitness(fs);
            fs->trialCount = 0;
        }
    }
}

void runABC(thrust::device_vector<FoodSource>& d_foodSources, hiprandState* state) {
    dim3 block(BLOCK_SIZE);
    dim3 grid((NUM_FOOD_SOURCES + block.x - 1) / block.x);
    for (int iter = 0; iter < MAX_ITERATIONS; iter++) {
        sendEmployedBees<<<grid, block>>>(thrust::raw_pointer_cast(d_foodSources.data()), state);
        hipDeviceSynchronize();
        sendOnlookerBees<<<grid, block>>>(thrust::raw_pointer_cast(d_foodSources.data()), state);
        hipDeviceSynchronize();
        sendScoutBees<<<grid, block>>>(thrust::raw_pointer_cast(d_foodSources.data()), state);
        hipDeviceSynchronize();
    }
}

void printResults(FoodSource* foodSources, double executionTime) {
    std::cout << std::fixed << std::setprecision(10);
    FoodSource bestFoodSource = foodSources[0];
    for (int i = 1; i < NUM_FOOD_SOURCES; i++) {
        if (foodSources[i].fitness < bestFoodSource.fitness) {
            bestFoodSource = foodSources[i];
        }
    }
    if (DIMENSIONS == 1) {
        std::cout << "Best Food Source Position: " << bestFoodSource.position[0] << std::endl;
    } else {
        std::cout << "Best Food Source Position: (";
        for (int i = 0; i < DIMENSIONS; i++) {
            std::cout << bestFoodSource.position[i];
            if (i < DIMENSIONS - 1) {
                std::cout << ", ";
            }
        }
        std::cout << ")" << std::endl;
    }
    std::cout << "Best Food Source Fitness: " << bestFoodSource.fitness << std::endl;
    std::cout << std::fixed << std::setprecision(2);
    std::cout << "Execution Time: " << executionTime << " milliseconds" << std::endl;
}

int main() {
    thrust::device_vector<FoodSource> d_foodSources(NUM_FOOD_SOURCES);
    hiprandState* state;
    hipMalloc(&state, NUM_FOOD_SOURCES * sizeof(hiprandState));
    auto start = std::chrono::high_resolution_clock::now();
    initializeFoodSources<<<(NUM_FOOD_SOURCES + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(thrust::raw_pointer_cast(d_foodSources.data()), state);
    hipDeviceSynchronize();
    runABC(d_foodSources, state);
    auto end = std::chrono::high_resolution_clock::now();
    double executionTime = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    thrust::host_vector<FoodSource> h_foodSources = d_foodSources;
    printResults(thrust::raw_pointer_cast(h_foodSources.data()), executionTime);
    hipFree(state);
    return 0;
}
