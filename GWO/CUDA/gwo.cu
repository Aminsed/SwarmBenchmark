#include "hip/hip_runtime.h"
#include "ObjectiveFunction.cuh"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <chrono>
#include <iostream>
#include <iomanip>
#include <fstream>

struct GreyWolf {
    double position[DIMENSIONS];
    double fitness;
};

__global__ void initializeGreyWolves(GreyWolf* wolves, double* alpha, double* beta, double* delta, hiprandState* state) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < NUM_WOLVES) {
        GreyWolf* wolf = &wolves[tid];
        hiprandState* s = &state[tid];
        hiprand_init(clock64(), tid, 0, s);
        for (int i = 0; i < DIMENSIONS; i++) {
            wolf->position[i] = hiprand_uniform_double(s) * 10.0 - 5.0;
        }
        wolf->fitness = objectiveFunction(wolf->position);
        if (wolf->fitness < alpha[DIMENSIONS]) {
            for (int i = 0; i < DIMENSIONS; i++) {
                alpha[i] = wolf->position[i];
            }
            alpha[DIMENSIONS] = wolf->fitness;
        }
        if (wolf->fitness < beta[DIMENSIONS] && wolf->fitness > alpha[DIMENSIONS]) {
            for (int i = 0; i < DIMENSIONS; i++) {
                beta[i] = wolf->position[i];
            }
            beta[DIMENSIONS] = wolf->fitness;
        }
        if (wolf->fitness < delta[DIMENSIONS] && wolf->fitness > beta[DIMENSIONS]) {
            for (int i = 0; i < DIMENSIONS; i++) {
                delta[i] = wolf->position[i];
            }
            delta[DIMENSIONS] = wolf->fitness;
        }
    }
}

__global__ void updateGreyWolves(GreyWolf* wolves, double* alpha, double* beta, double* delta, hiprandState* state, int iter) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < NUM_WOLVES) {
        GreyWolf* wolf = &wolves[tid];
        hiprandState* s = &state[tid];
        double a = 2.0 - (double)iter / MAX_ITERATIONS * 2.0;
        for (int i = 0; i < DIMENSIONS; i++) {
            double r1 = hiprand_uniform_double(s);
            double r2 = hiprand_uniform_double(s);
            double A1 = 2.0 * a * r1 - a;
            double C1 = 2.0 * r2;
            double D_alpha = abs(C1 * alpha[i] - wolf->position[i]);
            double X1 = alpha[i] - A1 * D_alpha;
            r1 = hiprand_uniform_double(s);
            r2 = hiprand_uniform_double(s);
            double A2 = 2.0 * a * r1 - a;
            double C2 = 2.0 * r2;
            double D_beta = abs(C2 * beta[i] - wolf->position[i]);
            double X2 = beta[i] - A2 * D_beta;
            r1 = hiprand_uniform_double(s);
            r2 = hiprand_uniform_double(s);
            double A3 = 2.0 * a * r1 - a;
            double C3 = 2.0 * r2;
            double D_delta = abs(C3 * delta[i] - wolf->position[i]);
            double X3 = delta[i] - A3 * D_delta;
            wolf->position[i] = (X1 + X2 + X3) / 3.0;
        }
        wolf->fitness = objectiveFunction(wolf->position);
        if (wolf->fitness < alpha[DIMENSIONS]) {
            for (int i = 0; i < DIMENSIONS; i++) {
                alpha[i] = wolf->position[i];
            }
            alpha[DIMENSIONS] = wolf->fitness;
        }
        if (wolf->fitness < beta[DIMENSIONS] && wolf->fitness > alpha[DIMENSIONS]) {
            for (int i = 0; i < DIMENSIONS; i++) {
                beta[i] = wolf->position[i];
            }
            beta[DIMENSIONS] = wolf->fitness;
        }
        if (wolf->fitness < delta[DIMENSIONS] && wolf->fitness > beta[DIMENSIONS]) {
            for (int i = 0; i < DIMENSIONS; i++) {
                delta[i] = wolf->position[i];
            }
            delta[DIMENSIONS] = wolf->fitness;
        }
    }
}


void runGWO(GreyWolf* wolves, double* alpha, double* beta, double* delta, hiprandState* state) {
    dim3 block(BLOCK_SIZE);
    dim3 grid((NUM_WOLVES + block.x - 1) / block.x);
    std::ofstream outputFile("results.txt");
    for (int iter = 0; iter < MAX_ITERATIONS; iter++) {
        updateGreyWolves<<<grid, block>>>(wolves, alpha, beta, delta, state, iter);
        hipDeviceSynchronize();
        double bestFitness;
        hipMemcpy(&bestFitness, &alpha[DIMENSIONS], sizeof(double), hipMemcpyDeviceToHost);
        outputFile << iter + 1 << ": " << bestFitness << std::endl;
    }
    outputFile.close();
}

void printResults(double* alpha, double executionTime) {
    std::cout << std::fixed << std::setprecision(10);
    if (DIMENSIONS == 1) {
        std::cout << "Best Position: " << alpha[0] << std::endl;
    } else {
        std::cout << "Best Position: (";
        for (int i = 0; i < DIMENSIONS; i++) {
            std::cout << alpha[i];
            if (i < DIMENSIONS - 1) {
                std::cout << ", ";
            }
        }
        std::cout << ")" << std::endl;
    }
    std::cout << "Best Fitness: " << alpha[DIMENSIONS] << std::endl;
    std::cout << std::fixed << std::setprecision(2);
    std::cout << "Execution Time: " << executionTime << " milliseconds" << std::endl;
}

int main() {
    GreyWolf* wolves;
    double* alpha;
    double* beta;
    double* delta;
    hiprandState* state;
    hipMalloc(&wolves, NUM_WOLVES * sizeof(GreyWolf));
    hipMalloc(&alpha, (DIMENSIONS + 1) * sizeof(double));
    hipMalloc(&beta, (DIMENSIONS + 1) * sizeof(double));
    hipMalloc(&delta, (DIMENSIONS + 1) * sizeof(double));
    hipMalloc(&state, NUM_WOLVES * sizeof(hiprandState));
    double initialFitness = INFINITY;
    hipMemcpy(&alpha[DIMENSIONS], &initialFitness, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(&beta[DIMENSIONS], &initialFitness, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(&delta[DIMENSIONS], &initialFitness, sizeof(double), hipMemcpyHostToDevice);

    auto start = std::chrono::high_resolution_clock::now();
    initializeGreyWolves<<<(NUM_WOLVES + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(wolves, alpha, beta, delta, state);
    hipDeviceSynchronize();

    std::ofstream outputFile("results.txt");
    for (int iter = 0; iter < MAX_ITERATIONS; iter++) {
        updateGreyWolves<<<(NUM_WOLVES + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(wolves, alpha, beta, delta, state, iter);
        hipDeviceSynchronize();

        double bestFitness;
        hipMemcpy(&bestFitness, &alpha[DIMENSIONS], sizeof(double), hipMemcpyDeviceToHost);
        outputFile << iter + 1 << ": " << bestFitness << std::endl;
    }
    outputFile.close();

    auto end = std::chrono::high_resolution_clock::now();
    double executionTime = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();

    double hostAlpha[DIMENSIONS + 1];
    hipMemcpy(hostAlpha, alpha, (DIMENSIONS + 1) * sizeof(double), hipMemcpyDeviceToHost);
    printResults(hostAlpha, executionTime);

    hipFree(wolves);
    hipFree(alpha);
    hipFree(beta);
    hipFree(delta);
    hipFree(state);
    hipDeviceReset();

    return 0;
}