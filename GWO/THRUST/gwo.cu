#include "hip/hip_runtime.h"
#include "ObjectiveFunction.cuh"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <chrono>
#include <iostream>
#include <iomanip>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <fstream>

struct GreyWolf {
    double position[DIMENSIONS];
    double fitness;
};

__device__ void updateLeaders(GreyWolf* wolf, double* alpha, double* beta, double* delta) {
    if (wolf->fitness < alpha[DIMENSIONS]) {
        for (int i = 0; i < DIMENSIONS; i++) {
            alpha[i] = wolf->position[i];
        }
        alpha[DIMENSIONS] = wolf->fitness;
    }
    if (wolf->fitness < beta[DIMENSIONS] && wolf->fitness > alpha[DIMENSIONS]) {
        for (int i = 0; i < DIMENSIONS; i++) {
            beta[i] = wolf->position[i];
        }
        beta[DIMENSIONS] = wolf->fitness;
    }
    if (wolf->fitness < delta[DIMENSIONS] && wolf->fitness > beta[DIMENSIONS]) {
        for (int i = 0; i < DIMENSIONS; i++) {
            delta[i] = wolf->position[i];
        }
        delta[DIMENSIONS] = wolf->fitness;
    }
}

__global__ void initializeGreyWolves(GreyWolf* wolves, double* alpha, double* beta, double* delta, hiprandState* state) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < NUM_WOLVES) {
        GreyWolf* wolf = &wolves[tid];
        hiprandState* s = &state[tid];
        hiprand_init(clock64(), tid, 0, s);
        for (int i = 0; i < DIMENSIONS; i++) {
            wolf->position[i] = hiprand_uniform_double(s) * 10.0 - 5.0;
        }
        wolf->fitness = objectiveFunction(wolf->position);
        updateLeaders(wolf, alpha, beta, delta);
    }
}

__global__ void updateGreyWolves(GreyWolf* wolves, double* alpha, double* beta, double* delta, hiprandState* state, int iter) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < NUM_WOLVES) {
        GreyWolf* wolf = &wolves[tid];
        hiprandState* s = &state[tid];
        double a = 2.0 - (double)iter / MAX_ITERATIONS * 2.0;
        for (int i = 0; i < DIMENSIONS; i++) {
            double r1 = hiprand_uniform_double(s);
            double r2 = hiprand_uniform_double(s);
            double A1 = 2.0 * a * r1 - a;
            double C1 = 2.0 * r2;
            double D_alpha = abs(C1 * alpha[i] - wolf->position[i]);
            double X1 = alpha[i] - A1 * D_alpha;
            r1 = hiprand_uniform_double(s);
            r2 = hiprand_uniform_double(s);
            double A2 = 2.0 * a * r1 - a;
            double C2 = 2.0 * r2;
            double D_beta = abs(C2 * beta[i] - wolf->position[i]);
            double X2 = beta[i] - A2 * D_beta;
            r1 = hiprand_uniform_double(s);
            r2 = hiprand_uniform_double(s);
            double A3 = 2.0 * a * r1 - a;
            double C3 = 2.0 * r2;
            double D_delta = abs(C3 * delta[i] - wolf->position[i]);
            double X3 = delta[i] - A3 * D_delta;
            wolf->position[i] = (X1 + X2 + X3) / 3.0;
        }
        wolf->fitness = objectiveFunction(wolf->position);
        updateLeaders(wolf, alpha, beta, delta);
    }
}

void runGWO(GreyWolf* wolves, double* alpha, double* beta, double* delta, hiprandState* state) {
    dim3 block(BLOCK_SIZE);
    dim3 grid((NUM_WOLVES + block.x - 1) / block.x);
    std::ofstream outputFile("results.txt");
    for (int iter = 0; iter < MAX_ITERATIONS; iter++) {
        updateGreyWolves<<<grid, block>>>(wolves, alpha, beta, delta, state, iter);
        hipDeviceSynchronize();
        double bestFitness;
        hipMemcpy(&bestFitness, &alpha[DIMENSIONS], sizeof(double), hipMemcpyDeviceToHost);
        outputFile << iter + 1 << ": " << bestFitness << std::endl;
    }
    outputFile.close();
}

void printResults(double* alpha, double executionTime) {
    std::cout << std::fixed << std::setprecision(10);
    if (DIMENSIONS == 1) {
        std::cout << "Best Position: " << alpha[0] << std::endl;
    } else {
        std::cout << "Best Position: (";
        for (int i = 0; i < DIMENSIONS; i++) {
            std::cout << alpha[i];
            if (i < DIMENSIONS - 1) {
                std::cout << ", ";
            }
        }
        std::cout << ")" << std::endl;
    }
    std::cout << "Best Fitness: " << alpha[DIMENSIONS] << std::endl;
    std::cout << std::fixed << std::setprecision(2);
    std::cout << "Execution Time: " << executionTime << " milliseconds" << std::endl;
}

int main() {
    thrust::device_vector<GreyWolf> d_wolves(NUM_WOLVES);
    thrust::device_vector<double> d_alpha(DIMENSIONS + 1);
    thrust::device_vector<double> d_beta(DIMENSIONS + 1);
    thrust::device_vector<double> d_delta(DIMENSIONS + 1);
    thrust::device_vector<hiprandState> d_state(NUM_WOLVES);

    double initialFitness = INFINITY;
    thrust::fill(d_alpha.begin() + DIMENSIONS, d_alpha.end(), initialFitness);
    thrust::fill(d_beta.begin() + DIMENSIONS, d_beta.end(), initialFitness);
    thrust::fill(d_delta.begin() + DIMENSIONS, d_delta.end(), initialFitness);

    auto start = std::chrono::high_resolution_clock::now();

    initializeGreyWolves<<<(NUM_WOLVES + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(
        thrust::raw_pointer_cast(d_wolves.data()),
        thrust::raw_pointer_cast(d_alpha.data()),
        thrust::raw_pointer_cast(d_beta.data()),
        thrust::raw_pointer_cast(d_delta.data()),
        thrust::raw_pointer_cast(d_state.data())
    );
    hipDeviceSynchronize();

    runGWO(
        thrust::raw_pointer_cast(d_wolves.data()),
        thrust::raw_pointer_cast(d_alpha.data()),
        thrust::raw_pointer_cast(d_beta.data()),
        thrust::raw_pointer_cast(d_delta.data()),
        thrust::raw_pointer_cast(d_state.data())
    );

    auto end = std::chrono::high_resolution_clock::now();
    double executionTime = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();

    double hostAlpha[DIMENSIONS + 1];
    hipMemcpy(hostAlpha, thrust::raw_pointer_cast(d_alpha.data()), (DIMENSIONS + 1) * sizeof(double), hipMemcpyDeviceToHost);
    printResults(hostAlpha, executionTime);

    return 0;
}