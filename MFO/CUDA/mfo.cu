#include "hip/hip_runtime.h"
#include "ObjectiveFunction.cuh"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <chrono>
#include <iostream>
#include <iomanip>
#include <cmath>
#include <fstream>

struct Moth {
    double position[DIMENSIONS];
    double fitness;
};

struct Flame {
    double position[DIMENSIONS];
};

__device__ void updateFlame(Moth* m, Flame* flame, double* bestFitness) {
    double fitness = objectiveFunction(m->position);
    if (fitness < objectiveFunction(flame->position)) {
        for (int i = 0; i < DIMENSIONS; i++) {
            flame->position[i] = m->position[i];
        }
    }
    // Update bestFitness without atomicMin
    if (fitness < *bestFitness) {
        *bestFitness = fitness;
    }
}


__global__ void initializeMoths(Moth* moths, Flame* flames, int* flameIndexes, hiprandState* state) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < NUM_MOTHS) {
        Moth* m = &moths[tid];
        Flame* f = &flames[tid];
        hiprandState* s = &state[tid];
        hiprand_init(clock64(), tid, 0, s);
        for (int i = 0; i < DIMENSIONS; i++) {
            m->position[i] = hiprand_uniform_double(s) * 10.0 - 5.0;
            f->position[i] = m->position[i];
        }
        m->fitness = objectiveFunction(m->position);
        flameIndexes[tid] = tid;
    }
}


__global__ void updateMoths(Moth* moths, Flame* flames, int* flameIndexes, hiprandState* state, int iter, double* bestFitness) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < NUM_MOTHS) {
        Moth* m = &moths[tid];
        hiprandState* s = &state[tid];
        int flameIndex = flameIndexes[tid];
        Flame* flame = &flames[flameIndex];
        for (int i = 0; i < DIMENSIONS; i++) {
            double t = static_cast<double>(iter) / MAX_ITERATIONS;
            double r = hiprand_uniform_double(s);
            double b = 1.0;
            double distance = fabs(flame->position[i] - m->position[i]);
            if (r < 0.5) {
                m->position[i] = distance * exp(b * t) * cos(t * 2 * M_PI) + flame->position[i];
            } else {
                m->position[i] = distance * exp(b * t) * sin(t * 2 * M_PI) + flame->position[i];
            }
        }
        m->fitness = objectiveFunction(m->position);
        updateFlame(m, flame, bestFitness);
    }
}

__global__ void sortMothsByFitness(Moth* moths, int* flameIndexes) {
    // Implement bubble sort (inefficient but simple)
    for (int i = 0; i < NUM_MOTHS - 1; i++) {
        for (int j = 0; j < NUM_MOTHS - i - 1; j++) {
            if (moths[j].fitness > moths[j + 1].fitness) {
                // Swap moths
                Moth temp = moths[j];
                moths[j] = moths[j + 1];
                moths[j + 1] = temp;

                // Swap flame indexes
                int tempIndex = flameIndexes[j];
                flameIndexes[j] = flameIndexes[j + 1];
                flameIndexes[j + 1] = tempIndex;
            }
        }
    }
}

void runMFO(Moth* moths, Flame* flames, int* flameIndexes, hiprandState* state, double* bestFitness) {
    dim3 block(BLOCK_SIZE);
    dim3 grid((NUM_MOTHS + block.x - 1) / block.x);
    std::ofstream outputFile("results.txt");
    for (int iter = 0; iter < MAX_ITERATIONS; iter++) {
        updateMoths<<<grid, block>>>(moths, flames, flameIndexes, state, iter, bestFitness);
        hipDeviceSynchronize();
        sortMothsByFitness<<<1, 1>>>(moths, flameIndexes);
        hipDeviceSynchronize();
        double currentBestFitness;
        hipMemcpy(&currentBestFitness, bestFitness, sizeof(double), hipMemcpyDeviceToHost);
        outputFile << iter + 1 << ": " << currentBestFitness << std::endl;
    }
    outputFile.close();
}

void printResults(Flame* flames, double* bestFitness, double executionTime) {
    std::cout << std::fixed << std::setprecision(10);
    if (DIMENSIONS == 1) {
        std::cout << "Best Flame Position: " << flames[0].position[0] << std::endl;
    } else {
        std::cout << "Best Flame Position: (";
        for (int i = 0; i < DIMENSIONS; i++) {
            std::cout << flames[0].position[i];
            if (i < DIMENSIONS - 1) {
                std::cout << ", ";
            }
        }
        std::cout << ")" << std::endl;
    }
    std::cout << "Best Flame Fitness: " << *bestFitness << std::endl;
    std::cout << std::fixed << std::setprecision(2);
    std::cout << "Execution Time: " << executionTime << " milliseconds" << std::endl;
}

int main() {
    Moth* moths;
    Flame* flames;
    int* flameIndexes;
    hiprandState* state;
    double* bestFitness;

    hipMalloc(&moths, NUM_MOTHS * sizeof(Moth));
    hipMalloc(&flames, NUM_MOTHS * sizeof(Flame));
    hipMalloc(&flameIndexes, NUM_MOTHS * sizeof(int));
    hipMalloc(&state, NUM_MOTHS * sizeof(hiprandState));
    hipMalloc(&bestFitness, sizeof(double));

    double initialBestFitness = std::numeric_limits<double>::max();
    hipMemcpy(bestFitness, &initialBestFitness, sizeof(double), hipMemcpyHostToDevice);

    auto start = std::chrono::high_resolution_clock::now();

    initializeMoths<<<(NUM_MOTHS + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(moths, flames, flameIndexes, state);
    hipDeviceSynchronize();

    runMFO(moths, flames, flameIndexes, state, bestFitness);

    auto end = std::chrono::high_resolution_clock::now();
    double executionTime = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();

    Flame hostFlames[NUM_MOTHS];
    double hostBestFitness;

    hipMemcpy(hostFlames, flames, NUM_MOTHS * sizeof(Flame), hipMemcpyDeviceToHost);
    hipMemcpy(&hostBestFitness, bestFitness, sizeof(double), hipMemcpyDeviceToHost);

    printResults(hostFlames, &hostBestFitness, executionTime);

    hipFree(moths);
    hipFree(flames);
    hipFree(flameIndexes);
    hipFree(state);
    hipFree(bestFitness);

    hipDeviceReset();

    return 0;
}