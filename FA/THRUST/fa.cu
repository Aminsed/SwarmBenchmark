#include "hip/hip_runtime.h"
#include "ObjectiveFunction.cuh"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <chrono>
#include <iostream>
#include <iomanip>
#include <cmath>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>

struct Firefly {
    double position[DIMENSIONS];
    double brightness;
};

__device__ double attractiveness(double distance) {
    return BETA0 * exp(-GAMMA * distance * distance);
}

__global__ void initializeFireflies(Firefly* fireflies, double* globalBestPosition, double* globalBestFitness, hiprandState* state) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < NUM_FIREFLIES) {
        Firefly* f = &fireflies[tid];
        hiprandState* s = &state[tid];
        hiprand_init(clock64(), tid, 0, s);

        for (int i = 0; i < DIMENSIONS; i++) {
            f->position[i] = hiprand_uniform_double(s) * 10.0 - 5.0;
        }
        f->brightness = objectiveFunction(f->position);

        if (f->brightness < *globalBestFitness) {
            *globalBestFitness = f->brightness;
            for (int i = 0; i < DIMENSIONS; i++) {
                globalBestPosition[i] = f->position[i];
            }
        }
    }
}

__global__ void updateFireflies(Firefly* fireflies, double* globalBestPosition, double* globalBestFitness, hiprandState* state) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < NUM_FIREFLIES) {
        Firefly* f = &fireflies[tid];
        hiprandState* s = &state[tid];

        for (int i = 0; i < NUM_FIREFLIES; i++) {
            if (i != tid) {
                Firefly* otherFirefly = &fireflies[i];
                double distance = 0.0;
                for (int j = 0; j < DIMENSIONS; j++) {
                    double diff = f->position[j] - otherFirefly->position[j];
                    distance += diff * diff;
                }
                distance = sqrt(distance);

                if (otherFirefly->brightness > f->brightness) {
                    double beta = attractiveness(distance);
                    for (int j = 0; j < DIMENSIONS; j++) {
                        double r = hiprand_uniform_double(s);
                        f->position[j] += beta * (otherFirefly->position[j] - f->position[j]) + ALPHA * (r - 0.5);
                    }
                }
            }
        }

        f->brightness = objectiveFunction(f->position);

        if (f->brightness < *globalBestFitness) {
            *globalBestFitness = f->brightness;
            for (int i = 0; i < DIMENSIONS; i++) {
                globalBestPosition[i] = f->position[i];
            }
        }
    }
}

void runFA(Firefly* fireflies, double* globalBestPosition, double* globalBestFitness, hiprandState* state) {
    dim3 block(BLOCK_SIZE);
    dim3 grid((NUM_FIREFLIES + block.x - 1) / block.x);

    for (int iter = 0; iter < MAX_ITERATIONS; iter++) {
        updateFireflies<<<grid, block>>>(fireflies, globalBestPosition, globalBestFitness, state);
        hipDeviceSynchronize();
    }
}

void printResults(double* globalBestPosition, double globalBestFitness, double executionTime) {
    std::cout << std::fixed << std::setprecision(10);
    if (DIMENSIONS == 1) {
        std::cout << "Global Best Position: " << globalBestPosition[0] << std::endl;
    } else {
        std::cout << "Global Best Position: (";
        for (int i = 0; i < DIMENSIONS; i++) {
            std::cout << globalBestPosition[i];
            if (i < DIMENSIONS - 1) {
                std::cout << ", ";
            }
        }
        std::cout << ")" << std::endl;
    }
    std::cout << "Global Best Value: " << globalBestFitness << std::endl;
    std::cout << std::fixed << std::setprecision(2);
    std::cout << "Execution Time: " << executionTime << " milliseconds" << std::endl;
}

int main() {
    thrust::device_vector<Firefly> fireflies(NUM_FIREFLIES);
    thrust::device_vector<double> globalBestPosition(DIMENSIONS);
    thrust::device_vector<double> globalBestFitness(1);
    thrust::device_vector<hiprandState> state(NUM_FIREFLIES);

    double initialFitness = INFINITY;
    globalBestFitness[0] = initialFitness;

    auto start = std::chrono::high_resolution_clock::now();

    initializeFireflies<<<(NUM_FIREFLIES + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(
        thrust::raw_pointer_cast(fireflies.data()),
        thrust::raw_pointer_cast(globalBestPosition.data()),
        thrust::raw_pointer_cast(globalBestFitness.data()),
        thrust::raw_pointer_cast(state.data())
    );
    hipDeviceSynchronize();

    runFA(
        thrust::raw_pointer_cast(fireflies.data()),
        thrust::raw_pointer_cast(globalBestPosition.data()),
        thrust::raw_pointer_cast(globalBestFitness.data()),
        thrust::raw_pointer_cast(state.data())
    );

    auto end = std::chrono::high_resolution_clock::now();
    double executionTime = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();

    double hostGlobalBestPosition[DIMENSIONS];
    double hostGlobalBestFitness = globalBestFitness[0];
    thrust::copy(globalBestPosition.begin(), globalBestPosition.end(), hostGlobalBestPosition);

    printResults(hostGlobalBestPosition, hostGlobalBestFitness, executionTime);

    return 0;
}
