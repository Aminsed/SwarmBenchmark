#include "hip/hip_runtime.h"
// GrasshopperOptimizationAlgorithm.cu
#include "ObjectiveFunction.cuh"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <chrono>
#include <iostream>
#include <iomanip>
#include <cmath>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/copy.h>
#include <fstream>

struct Grasshopper {
    double position[DIMENSIONS];
};

__device__ void updateBestFitness(Grasshopper* g, double* globalBestPosition, double* globalBestFitness) {
    double fitness = objectiveFunction(g->position);
    if (fitness < *globalBestFitness) {
        *globalBestFitness = fitness;
        for (int i = 0; i < DIMENSIONS; i++) {
            globalBestPosition[i] = g->position[i];
        }
    }
}

__global__ void initializeGrasshoppers(Grasshopper* grasshoppers, double* globalBestPosition, double* globalBestFitness, hiprandState* state) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < NUM_GRASSHOPPERS) {
        Grasshopper* g = &grasshoppers[tid];
        hiprandState* s = &state[tid];
        hiprand_init(clock64(), tid, 0, s);
        for (int i = 0; i < DIMENSIONS; i++) {
            g->position[i] = hiprand_uniform_double(s) * (UPPER_BOUND - LOWER_BOUND) + LOWER_BOUND;
        }
        updateBestFitness(g, globalBestPosition, globalBestFitness);
    }
}

__global__ void updateGrasshoppers(Grasshopper* grasshoppers, double* globalBestPosition, double* globalBestFitness, hiprandState* state, int iter) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < NUM_GRASSHOPPERS) {
        Grasshopper* g = &grasshoppers[tid];
        hiprandState* s = &state[tid];
        double c = 0.00001;
        double cMax = 1.0;
        double cMin = 0.00001;
        double l = (cMax - cMin) * (MAX_ITERATIONS - iter) / MAX_ITERATIONS + cMin;
        for (int i = 0; i < DIMENSIONS; i++) {
            double socialInteraction = 0.0;
            for (int j = 0; j < NUM_GRASSHOPPERS; j++) {
                if (j != tid) {
                    double distance = fabs(g->position[i] - grasshoppers[j].position[i]);
                    double r = hiprand_uniform_double(s);
                    double si = (0.5 + 0.5 * r) * (globalBestPosition[i] - l * distance);
                    socialInteraction += si;
                }
            }
            double xi = c * socialInteraction;
            double r = hiprand_uniform_double(s);
            double levy = pow(r, -1.0 / LEVY_EXPONENT);
            double newPosition = g->position[i] + xi * levy;
            g->position[i] = fmax(LOWER_BOUND, fmin(newPosition, UPPER_BOUND));
        }
        updateBestFitness(g, globalBestPosition, globalBestFitness);
    }
}

void runGOA(thrust::device_vector<Grasshopper>& grasshoppers, thrust::device_vector<double>& globalBestPosition, thrust::device_vector<double>& globalBestFitness, thrust::device_vector<hiprandState>& state) {
    std::ofstream outputFile("results.txt");
    dim3 block(BLOCK_SIZE);
    dim3 grid((NUM_GRASSHOPPERS + block.x - 1) / block.x);
    for (int iter = 0; iter < MAX_ITERATIONS; iter++) {
        updateGrasshoppers<<<grid, block>>>(thrust::raw_pointer_cast(grasshoppers.data()), thrust::raw_pointer_cast(globalBestPosition.data()), thrust::raw_pointer_cast(globalBestFitness.data()), thrust::raw_pointer_cast(state.data()), iter);
        hipDeviceSynchronize();
        double hostGlobalBestFitness = globalBestFitness[0];
        outputFile << iter + 1 << ": " << hostGlobalBestFitness << std::endl;
    }
    outputFile.close();
}

void printResults(thrust::host_vector<double>& globalBestPosition, double globalBestFitness, double executionTime) {
    std::cout << std::fixed << std::setprecision(10);
    if (DIMENSIONS == 1) {
        std::cout << "Global Best Position: " << globalBestPosition[0] << std::endl;
    } else {
        std::cout << "Global Best Position: (";
        for (int i = 0; i < DIMENSIONS; i++) {
            std::cout << globalBestPosition[i];
            if (i < DIMENSIONS - 1) {
                std::cout << ", ";
            }
        }
        std::cout << ")" << std::endl;
    }
    std::cout << "Global Best Value: " << globalBestFitness << std::endl;
    std::cout << std::fixed << std::setprecision(2);
    std::cout << "Execution Time: " << executionTime << " milliseconds" << std::endl;
}

int main() {
    thrust::device_vector<Grasshopper> grasshoppers(NUM_GRASSHOPPERS);
    thrust::device_vector<double> globalBestPosition(DIMENSIONS);
    thrust::device_vector<double> globalBestFitness(1);
    thrust::device_vector<hiprandState> state(NUM_GRASSHOPPERS);
    double initialFitness = INFINITY;
    thrust::copy(&initialFitness, &initialFitness + 1, globalBestFitness.begin());
    auto start = std::chrono::high_resolution_clock::now();
    initializeGrasshoppers<<<(NUM_GRASSHOPPERS + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(thrust::raw_pointer_cast(grasshoppers.data()), thrust::raw_pointer_cast(globalBestPosition.data()), thrust::raw_pointer_cast(globalBestFitness.data()), thrust::raw_pointer_cast(state.data()));
    hipDeviceSynchronize();
    runGOA(grasshoppers, globalBestPosition, globalBestFitness, state);
    auto end = std::chrono::high_resolution_clock::now();
    double executionTime = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    thrust::host_vector<double> hostGlobalBestPosition(DIMENSIONS);
    thrust::copy(globalBestPosition.begin(), globalBestPosition.end(), hostGlobalBestPosition.begin());
    double hostGlobalBestFitness = globalBestFitness[0];
    printResults(hostGlobalBestPosition, hostGlobalBestFitness, executionTime);
    return 0;
}
