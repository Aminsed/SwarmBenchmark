#include "hip/hip_runtime.h"
#include "ObjectiveFunction.cuh"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <chrono>
#include <iostream>
#include <iomanip>
#include <cmath>

struct Grasshopper {
    double position[DIMENSIONS];
};

__device__ void updateBestFitness(Grasshopper* g, double* globalBestPosition, double* globalBestFitness) {
    double fitness = objectiveFunction(g->position);
    if (fitness < *globalBestFitness) {
        *globalBestFitness = fitness;
        for (int i = 0; i < DIMENSIONS; i++) {
            globalBestPosition[i] = g->position[i];
        }
    }
}

__global__ void initializeGrasshoppers(Grasshopper* grasshoppers, double* globalBestPosition, double* globalBestFitness, hiprandState* state) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < NUM_GRASSHOPPERS) {
        Grasshopper* g = &grasshoppers[tid];
        hiprandState* s = &state[tid];
        hiprand_init(clock64(), tid, 0, s);
        for (int i = 0; i < DIMENSIONS; i++) {
            g->position[i] = hiprand_uniform_double(s) * 10.0 - 5.0;
        }
        updateBestFitness(g, globalBestPosition, globalBestFitness);
    }
}

__global__ void updateGrasshoppers(Grasshopper* grasshoppers, double* globalBestPosition, double* globalBestFitness, hiprandState* state, int iter) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < NUM_GRASSHOPPERS) {
        Grasshopper* g = &grasshoppers[tid];
        hiprandState* s = &state[tid];
        double c = 0.00001;
        double cMax = 1.0;
        double cMin = 0.00001;
        double l = (cMax - cMin) * (MAX_ITERATIONS - iter) / MAX_ITERATIONS + cMin;
        for (int i = 0; i < DIMENSIONS; i++) {
            double socialInteraction = 0.0;
            for (int j = 0; j < NUM_GRASSHOPPERS; j++) {
                if (j != tid) {
                    double distance = fabs(g->position[i] - grasshoppers[j].position[i]);
                    double r = hiprand_uniform_double(s);
                    double si = (0.5 + 0.5 * r) * (globalBestPosition[i] - l * distance);
                    socialInteraction += si;
                }
            }
            double xi = c * socialInteraction;
            double r = hiprand_uniform_double(s);
            double levy = pow(r, -1.0 / LEVY_EXPONENT);
            double newPosition = g->position[i] + xi * levy;
            g->position[i] = fmax(LOWER_BOUND, fmin(newPosition, UPPER_BOUND));
        }
        updateBestFitness(g, globalBestPosition, globalBestFitness);
    }
}

void runGOA(Grasshopper* grasshoppers, double* globalBestPosition, double* globalBestFitness, hiprandState* state) {
    dim3 block(BLOCK_SIZE);
    dim3 grid((NUM_GRASSHOPPERS + block.x - 1) / block.x);
    for (int iter = 0; iter < MAX_ITERATIONS; iter++) {
        updateGrasshoppers<<<grid, block>>>(grasshoppers, globalBestPosition, globalBestFitness, state, iter);
        hipDeviceSynchronize();
    }
}

void printResults(double* globalBestPosition, double globalBestFitness, double executionTime) {
    std::cout << std::fixed << std::setprecision(10);
    if (DIMENSIONS == 1) {
        std::cout << "Global Best Position: " << globalBestPosition[0] << std::endl;
    } else {
        std::cout << "Global Best Position: (";
        for (int i = 0; i < DIMENSIONS; i++) {
            std::cout << globalBestPosition[i];
            if (i < DIMENSIONS - 1) {
                std::cout << ", ";
            }
        }
        std::cout << ")" << std::endl;
    }
    std::cout << "Global Best Value: " << globalBestFitness << std::endl;
    std::cout << std::fixed << std::setprecision(2);
    std::cout << "Execution Time: " << executionTime << " milliseconds" << std::endl;
}

int main() {
    Grasshopper* grasshoppers;
    double* globalBestPosition;
    double* globalBestFitness;
    hiprandState* state;
    hipMalloc(&grasshoppers, NUM_GRASSHOPPERS * sizeof(Grasshopper));
    hipMalloc(&globalBestPosition, DIMENSIONS * sizeof(double));
    hipMalloc(&globalBestFitness, sizeof(double));
    hipMalloc(&state, NUM_GRASSHOPPERS * sizeof(hiprandState));
    double initialFitness = INFINITY;
    hipMemcpy(globalBestFitness, &initialFitness, sizeof(double), hipMemcpyHostToDevice);
    auto start = std::chrono::high_resolution_clock::now();
    initializeGrasshoppers<<<(NUM_GRASSHOPPERS + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(grasshoppers, globalBestPosition, globalBestFitness, state);
    hipDeviceSynchronize();
    runGOA(grasshoppers, globalBestPosition, globalBestFitness, state);
    auto end = std::chrono::high_resolution_clock::now();
    double executionTime = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    double hostGlobalBestPosition[DIMENSIONS];
    double hostGlobalBestFitness;
    hipMemcpy(hostGlobalBestPosition, globalBestPosition, DIMENSIONS * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(&hostGlobalBestFitness, globalBestFitness, sizeof(double), hipMemcpyDeviceToHost);
    printResults(hostGlobalBestPosition, hostGlobalBestFitness, executionTime);
    hipFree(grasshoppers);
    hipFree(globalBestPosition);
    hipFree(globalBestFitness);
    hipFree(state);
    hipDeviceReset();
    return 0;
}
