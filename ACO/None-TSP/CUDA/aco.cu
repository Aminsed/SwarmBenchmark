#include "hip/hip_runtime.h"
#include "ObjectiveFunction.cuh"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <chrono>
#include <iostream>
#include <iomanip>
#include <fstream>


struct Ant {
    double position[DIMENSIONS];
    double fitness;
};

__device__ void updatePheromone(double* pheromone, double* bestPosition, double bestFitness) {
    for (int i = 0; i < DIMENSIONS; i++) {
        pheromone[i] += Q / (bestFitness + 1e-10);
    }
}

__global__ void initializeAnts(Ant* ants, double* pheromone, hiprandState* state, unsigned long long seed) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < NUM_ANTS) {
        hiprand_init(seed, tid, 0, &state[tid]);
        Ant* a = &ants[tid];
        for (int i = 0; i < DIMENSIONS; i++) {
            a->position[i] = hiprand_uniform_double(&state[tid]) * 10.0 - 5.0;
            pheromone[i] = 1.0;
        }
        a->fitness = objectiveFunction(a->position);
    }
}

__global__ void updateAnts(Ant* ants, double* pheromone, double* bestPosition, double* bestFitness, hiprandState* state) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < NUM_ANTS) {
        Ant* a = &ants[tid];
        for (int i = 0; i < DIMENSIONS; i++) {
            double r = hiprand_uniform_double(&state[tid]);
            if (r < PHEROMONE_WEIGHT) {
                a->position[i] = bestPosition[i] + (hiprand_uniform_double(&state[tid]) * 2.0 - 1.0);
            } else {
                a->position[i] += hiprand_uniform_double(&state[tid]) * 2.0 - 1.0;
            }
        }
        a->fitness = objectiveFunction(a->position);
        
        // Update best fitness and position
        if (a->fitness < *bestFitness) {
            *bestFitness = a->fitness;
            for (int i = 0; i < DIMENSIONS; i++) {
                bestPosition[i] = a->position[i];
            }
        }
    }
}


void runACO(Ant* ants, double* pheromone, double* bestPosition, double* bestFitness, hiprandState* state) {
    std::ofstream outputFile("results.txt");
    dim3 block(BLOCK_SIZE);
    dim3 grid((NUM_ANTS + block.x - 1) / block.x);
    for (int iter = 0; iter < MAX_ITERATIONS; iter++) {
        updateAnts<<<grid, block>>>(ants, pheromone, bestPosition, bestFitness, state);
        hipDeviceSynchronize();
        double hostBestFitness;
        hipMemcpy(&hostBestFitness, bestFitness, sizeof(double), hipMemcpyDeviceToHost);
        outputFile << iter + 1 << ": " << hostBestFitness << std::endl;
    }
    outputFile.close();
}

void printResults(double* bestPosition, double bestFitness, double executionTime) {
    std::cout << std::fixed << std::setprecision(10);
    if (DIMENSIONS == 1) {
        std::cout << "Best Position: " << bestPosition[0] << std::endl;
    } else {
        std::cout << "Best Position: (";
        for (int i = 0; i < DIMENSIONS; i++) {
            std::cout << bestPosition[i];
            if (i < DIMENSIONS - 1) {
                std::cout << ", ";
            }
        }
        std::cout << ")" << std::endl;
    }
    std::cout << "Best Value: " << bestFitness << std::endl;
    std::cout << std::fixed << std::setprecision(2);
    std::cout << "Execution Time: " << executionTime << " milliseconds" << std::endl;
}

int main() {
    Ant* ants;
    double* pheromone;
    double* bestPosition;
    double* bestFitness;
    hiprandState* state;
    
    hipMalloc(&ants, NUM_ANTS * sizeof(Ant));
    hipMalloc(&pheromone, DIMENSIONS * sizeof(double));
    hipMalloc(&bestPosition, DIMENSIONS * sizeof(double));
    hipMalloc(&bestFitness, sizeof(double));
    hipMalloc(&state, NUM_ANTS * sizeof(hiprandState));
    
    double initialFitness = INFINITY;
    hipMemcpy(bestFitness, &initialFitness, sizeof(double), hipMemcpyHostToDevice);
    
    auto start = std::chrono::high_resolution_clock::now();
    
    unsigned long long seed = time(NULL);
    dim3 block(BLOCK_SIZE);
    dim3 grid((NUM_ANTS + block.x - 1) / block.x);
    
    initializeAnts<<<grid, block>>>(ants, pheromone, state, seed);
    hipDeviceSynchronize();
    
    runACO(ants, pheromone, bestPosition, bestFitness, state);
    
    auto end = std::chrono::high_resolution_clock::now();
    double executionTime = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    
    double hostBestPosition[DIMENSIONS];
    double hostBestFitness;
    hipMemcpy(hostBestPosition, bestPosition, DIMENSIONS * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(&hostBestFitness, bestFitness, sizeof(double), hipMemcpyDeviceToHost);
    
    printResults(hostBestPosition, hostBestFitness, executionTime);
    
    hipFree(ants);
    hipFree(pheromone);
    hipFree(bestPosition);
    hipFree(bestFitness);
    hipFree(state);
    hipDeviceReset();
    
    return 0;
}
