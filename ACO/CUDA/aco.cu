#include "hip/hip_runtime.h"
#include "ObjectiveFunction.cuh"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <chrono>
#include <iostream>
#include <iomanip>

struct Ant {
    double position[DIMENSIONS];
    double fitness;
};

__device__ void updatePheromone(double* pheromone, double* bestPosition, double bestFitness) {
    for (int i = 0; i < DIMENSIONS; i++) {
        pheromone[i] += Q / bestFitness;
    }
}

__global__ void initializeAnts(Ant* ants, double* pheromone, hiprandState* state) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < NUM_ANTS) {
        Ant* a = &ants[tid];
        hiprandState* s = &state[tid];
        hiprand_init(clock64(), tid, 0, s);
        for (int i = 0; i < DIMENSIONS; i++) {
            a->position[i] = hiprand_uniform_double(s) * 10.0 - 5.0;
        }
        a->fitness = objectiveFunction(a->position);
    }
}

__global__ void updateAnts(Ant* ants, double* pheromone, double* bestPosition, double* bestFitness, hiprandState* state) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < NUM_ANTS) {
        Ant* a = &ants[tid];
        hiprandState* s = &state[tid];
        for (int i = 0; i < DIMENSIONS; i++) {
            double r = hiprand_uniform_double(s);
            if (r < PHEROMONE_WEIGHT) {
                a->position[i] = bestPosition[i];
            } else {
                a->position[i] += hiprand_uniform_double(s) * 2.0 - 1.0;
            }
        }
        a->fitness = objectiveFunction(a->position);
        if (a->fitness < *bestFitness) {
            *bestFitness = a->fitness;
            for (int i = 0; i < DIMENSIONS; i++) {
                bestPosition[i] = a->position[i];
            }
            updatePheromone(pheromone, bestPosition, *bestFitness);
        }
    }
}

void runACO(Ant* ants, double* pheromone, double* bestPosition, double* bestFitness, hiprandState* state) {
    dim3 block(BLOCK_SIZE);
    dim3 grid((NUM_ANTS + block.x - 1) / block.x);
    for (int iter = 0; iter < MAX_ITERATIONS; iter++) {
        updateAnts<<<grid, block>>>(ants, pheromone, bestPosition, bestFitness, state);
        hipDeviceSynchronize();
    }
}

void printResults(double* bestPosition, double bestFitness, double executionTime) {
    std::cout << std::fixed << std::setprecision(10);
    if (DIMENSIONS == 1) {
        std::cout << "Best Position: " << bestPosition[0] << std::endl;
    } else {
        std::cout << "Best Position: (";
        for (int i = 0; i < DIMENSIONS; i++) {
            std::cout << bestPosition[i];
            if (i < DIMENSIONS - 1) {
                std::cout << ", ";
            }
        }
        std::cout << ")" << std::endl;
    }
    std::cout << "Best Value: " << bestFitness << std::endl;
    std::cout << std::fixed << std::setprecision(2);
    std::cout << "Execution Time: " << executionTime << " milliseconds" << std::endl;
}

int main() {
    Ant* ants;
    double* pheromone;
    double* bestPosition;
    double* bestFitness;
    hiprandState* state;
    hipMalloc(&ants, NUM_ANTS * sizeof(Ant));
    hipMalloc(&pheromone, DIMENSIONS * sizeof(double));
    hipMalloc(&bestPosition, DIMENSIONS * sizeof(double));
    hipMalloc(&bestFitness, sizeof(double));
    hipMalloc(&state, NUM_ANTS * sizeof(hiprandState));
    double initialFitness = INFINITY;
    hipMemcpy(bestFitness, &initialFitness, sizeof(double), hipMemcpyHostToDevice);
    auto start = std::chrono::high_resolution_clock::now();
    initializeAnts<<<(NUM_ANTS + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(ants, pheromone, state);
    hipDeviceSynchronize();
    runACO(ants, pheromone, bestPosition, bestFitness, state);
    auto end = std::chrono::high_resolution_clock::now();
    double executionTime = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    double hostBestPosition[DIMENSIONS];
    double hostBestFitness;
    hipMemcpy(hostBestPosition, bestPosition, DIMENSIONS * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(&hostBestFitness, bestFitness, sizeof(double), hipMemcpyDeviceToHost);
    printResults(hostBestPosition, hostBestFitness, executionTime);
    hipFree(ants);
    hipFree(pheromone);
    hipFree(bestPosition);
    hipFree(bestFitness);
    hipFree(state);
    hipDeviceReset();
    return 0;
}
