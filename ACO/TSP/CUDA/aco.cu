#include "hip/hip_runtime.h"
#include "ObjectiveFunction.cuh"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <chrono>
#include <iostream>
#include <iomanip>
#include <vector>
#include <algorithm>

struct Ant {
    int tour[NUM_CITIES];
    double tourLength;
};

__device__ double calculateTourLength(int* tour) {
    double length = 0.0;
    for (int i = 0; i < NUM_CITIES - 1; i++) {
        int city1 = tour[i];
        int city2 = tour[i + 1];
        length += distances[city1][city2];
    }
    int lastCity = tour[NUM_CITIES - 1];
    int firstCity = tour[0];
    length += distances[lastCity][firstCity];
    return length;
}

__global__ void constructTours(Ant* ants, double* pheromones, int* bestTour, double* bestTourLength, hiprandState* state) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < NUM_ANTS) {
        Ant* ant = &ants[tid];
        hiprandState* s = &state[tid];

        int visited[NUM_CITIES] = {0};
        int current = hiprand(s) % NUM_CITIES;
        ant->tour[0] = current;
        visited[current] = 1;

        for (int i = 1; i < NUM_CITIES; i++) {
            double probabilities[NUM_CITIES] = {0};
            double sum = 0.0;

            for (int j = 0; j < NUM_CITIES; j++) {
                if (!visited[j]) {
                    probabilities[j] = pow(pheromones[current * NUM_CITIES + j], ALPHA) *
                                       pow(1.0 / distances[current][j], BETA);
                    sum += probabilities[j];
                }
            }

            double r = hiprand_uniform_double(s) * sum;
            double cumulativeProb = 0.0;
            int nextCity = -1;

            for (int j = 0; j < NUM_CITIES; j++) {
                if (!visited[j]) {
                    cumulativeProb += probabilities[j];
                    if (r <= cumulativeProb) {
                        nextCity = j;
                        break;
                    }
                }
            }

            ant->tour[i] = nextCity;
            visited[nextCity] = 1;
            current = nextCity;
        }

        ant->tourLength = calculateTourLength(ant->tour);

        if (ant->tourLength < *bestTourLength) {
            *bestTourLength = ant->tourLength;
            for (int i = 0; i < NUM_CITIES; i++) {
                bestTour[i] = ant->tour[i];
            }
        }
    }
}

__global__ void updatePheromones(double* pheromones, Ant* ants) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < NUM_CITIES * NUM_CITIES) {
        int i = tid / NUM_CITIES;
        int j = tid % NUM_CITIES;
        pheromones[i * NUM_CITIES + j] *= EVAPORATION_RATE;

        for (int k = 0; k < NUM_ANTS; k++) {
            Ant* ant = &ants[k];
            for (int l = 0; l < NUM_CITIES - 1; l++) {
                if (ant->tour[l] == i && ant->tour[l + 1] == j) {
                    pheromones[i * NUM_CITIES + j] += Q / ant->tourLength;
                }
            }
            if (ant->tour[NUM_CITIES - 1] == i && ant->tour[0] == j) {
                pheromones[i * NUM_CITIES + j] += Q / ant->tourLength;
            }
        }
    }
}

void runACO(Ant* ants, double* pheromones, int* bestTour, double* bestTourLength, hiprandState* state) {
    dim3 block(BLOCK_SIZE);
    dim3 grid((NUM_ANTS + block.x - 1) / block.x);

    for (int iter = 0; iter < MAX_ITERATIONS; iter++) {
        constructTours<<<grid, block>>>(ants, pheromones, bestTour, bestTourLength, state);
        updatePheromones<<<(NUM_CITIES * NUM_CITIES + block.x - 1) / block.x, block>>>(pheromones, ants);
        hipDeviceSynchronize();
    }
}

void printResults(int* bestTour, double bestTourLength, double executionTime) {
    std::cout << "Best Tour: ";
    for (int i = 0; i < NUM_CITIES; i++) {
        std::cout << bestTour[i] << " ";
    }
    std::cout << std::endl;
    std::cout << "Best Tour Length: " << bestTourLength << std::endl;
    std::cout << std::fixed << std::setprecision(2);
    std::cout << "Execution Time: " << executionTime << " milliseconds" << std::endl;
}

int main() {
    Ant* ants;
    double* pheromones;
    int* bestTour;
    double* bestTourLength;
    hiprandState* state;

    hipMalloc(&ants, NUM_ANTS * sizeof(Ant));
    hipMalloc(&pheromones, NUM_CITIES * NUM_CITIES * sizeof(double));
    hipMalloc(&bestTour, NUM_CITIES * sizeof(int));
    hipMalloc(&bestTourLength, sizeof(double));
    hipMalloc(&state, NUM_ANTS * sizeof(hiprandState));

    double initialPheromone = 1.0;
    hipMemset(pheromones, initialPheromone, NUM_CITIES * NUM_CITIES * sizeof(double));

    double initialTourLength = INFINITY;
    hipMemcpy(bestTourLength, &initialTourLength, sizeof(double), hipMemcpyHostToDevice);

    auto start = std::chrono::high_resolution_clock::now();
    runACO(ants, pheromones, bestTour, bestTourLength, state);
    auto end = std::chrono::high_resolution_clock::now();
    double executionTime = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();

    int* hostBestTour = new int[NUM_CITIES];
    double hostBestTourLength;
    hipMemcpy(hostBestTour, bestTour, NUM_CITIES * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&hostBestTourLength, bestTourLength, sizeof(double), hipMemcpyDeviceToHost);

    printResults(hostBestTour, hostBestTourLength, executionTime);

    delete[] hostBestTour;
    hipFree(ants);
    hipFree(pheromones);
    hipFree(bestTour);
    hipFree(bestTourLength);
    hipFree(state);
    hipDeviceReset();

    return 0;
}
